#include <stdio.h>
#include <stdlib.h>
#include "bin_reader.h"
#include "timer.h"
#include "hip/hip_runtime.h"

int main(int argc, char ** argv) {

#if 0
    float * vectorA;
    binReadAsArray<float>("gold/vecA.bin", NULL, &vectorA);

    for (size_t idx=0;idx<1000;idx++)
        printf("%.4f\n", vectorA[idx]);
#else
    float * vectorA;
    BinInfo bi;
    if (binReadAsArrayNP<float>("gold/vecA.bin", &bi, &vectorA) == -1)
        printf("ERROR!\n");

    for (size_t idx=0;idx<1000;idx++)
        printf("%.4f\n", vectorA[idx]);

    void * memd;
    hipMalloc(&memd, bi.size);
    hipMemcpy(memd, vectorA, bi.size - bi.offset, hipMemcpyHostToDevice);

    binDiscardArrayNP(vectorA);
#endif

    return 0;
}

